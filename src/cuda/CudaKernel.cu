#include "hip/hip_runtime.h"

#include "CudaKernel.hpp"

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <stdio.h>

#include "../common/Timer.hpp"
#include "MatrixView.hpp"
#include "VectorView.hpp"

using namespace std;

static MatrixView uploadToDevice(MatrixView &mv);
static MatrixView viewFromMatrix(const EMatrix &mat);
static EMatrix matrixFromView(const MatrixView &view);
static MatrixView newView(unsigned rows, unsigned cols);
static void releaseView(MatrixView &view);

__global__ void multiplyKernel0(MatrixView a, MatrixView b, MatrixView out) {
  int outRow = blockDim.y * blockIdx.y + threadIdx.y;
  int outCol = blockDim.x * blockIdx.x + threadIdx.x;
  if (outRow >= out.rows || outCol >= out.cols) {
    return;
  }

  float r = 0.0f;
  for (int i = 0; i < a.cols; i++) {
    r += a.data[i + outRow * a.cols] * b.data[outCol + i * b.cols];
  }
  out.data[outCol + outRow * out.cols] = r;
}

// Use shared memory.
__global__ void multiplyKernel1(MatrixView a, MatrixView b, MatrixView out) {
  extern __shared__ float buf[];

  const int outRow = blockDim.y * blockIdx.y + threadIdx.y;
  const int outCol = blockDim.x * blockIdx.x + threadIdx.x;

  const int numChunks = (a.cols + blockDim.x - 1) / blockDim.x;

  float *aChunk = (float *) buf;
  float *bChunk = (float *) &buf[blockDim.x * blockDim.y];

  float value = 0.0f;
  for (int i = 0; i < numChunks; i++) {
    const int chunkOffset = i * blockDim.x;
    const int chunkIndex = threadIdx.x + threadIdx.y * blockDim.x;

    const int aRow = outRow;
    const int aCol = chunkOffset + threadIdx.x;

    const int bRow = chunkOffset + threadIdx.y;
    const int bCol = outCol;

    if (aCol < a.cols) {
      aChunk[chunkIndex] = a.data[aCol + aRow * a.cols];
    }
    if (bRow < b.rows) {
      bChunk[chunkIndex] = b.data[bCol + bRow * b.cols];
    }

    __syncthreads();

    if (outRow < out.rows && outCol < out.cols) {
      int chunkLim = min(blockDim.x, a.cols - chunkOffset);
      for (int j = 0; j < chunkLim; j++) {
        value += aChunk[j + threadIdx.y * blockDim.x] * bChunk[threadIdx.x + j * blockDim.x];
      }
    }
    __syncthreads();
  }

  if (outRow < out.rows && outCol < out.cols) {
    out.data[outCol + outRow * out.cols] = value;
  }
}

EMatrix CudaKernel::Multiply0(EMatrix &a, EMatrix &b) {
  MatrixView h_A = viewFromMatrix(a);
  MatrixView h_B = viewFromMatrix(b);
  MatrixView h_R = newView(h_A.rows, h_B.cols);

  MatrixView d_A = uploadToDevice(h_A);
  MatrixView d_B = uploadToDevice(h_B);

  MatrixView d_R = h_R;
  hipMalloc(&(d_R.data), h_R.rows * h_R.cols * sizeof(float));

  // Threads per block in X and Y dimensions.
  int tpbX = 32;
  int tpbY = 32;

  // Blocks per grid in X and Y dimensions.
  int bpgX = (h_R.cols + tpbX - 1) / tpbX;
  int bpgY = (h_R.rows + tpbY - 1) / tpbY;

    size_t sharedMemSize = 2 * tpbX * tpbY * sizeof(float);

  Timer timer;
  timer.Start();
  for (unsigned i = 0; i < 10; i++) {
    multiplyKernel1<<<dim3(bpgX, bpgY, 1), dim3(tpbX, tpbY, 1), sharedMemSize>>>(d_A, d_B, d_R);
  }
  hipMemcpy(h_R.data, d_R.data, h_R.rows * h_R.cols * sizeof(float), hipMemcpyDeviceToHost);

  timer.Stop();
  cout << "elapsed time: " << timer.GetNumElapsedMicroseconds() << endl;

  EMatrix result = matrixFromView(h_R);

  hipFree(d_R.data);
  hipFree(d_B.data);
  hipFree(d_A.data);

  releaseView(h_R);
  releaseView(h_B);
  releaseView(h_A);

  return result;
}

static MatrixView uploadToDevice(MatrixView &h_mv) {
  unsigned dataSize = h_mv.rows * h_mv.cols * sizeof(float);
  MatrixView d_mv = h_mv;
  hipMalloc(&(d_mv.data), dataSize);
  hipMemcpy(d_mv.data, h_mv.data, dataSize, hipMemcpyHostToDevice);
  return d_mv;
}

MatrixView viewFromMatrix(const EMatrix &mat) {
  MatrixView rv;
  rv.rows = static_cast<unsigned>(mat.rows());
  rv.cols = static_cast<unsigned>(mat.cols());
  rv.data = new float[rv.rows * rv.cols];

  unsigned i = 0;
  for (unsigned r = 0; r < rv.rows; r++) {
    for (unsigned c = 0; c < rv.cols; c++) {
      rv.data[i++] = mat(r, c);
    }
  }
  return rv;
}

EMatrix matrixFromView(const MatrixView &view) {
  EMatrix rm(view.rows, view.cols);
  unsigned i = 0;
  for (unsigned r = 0; r < view.rows; r++) {
    for (unsigned c = 0; c < view.cols; c++) {
      rm(r, c) = view.data[i++];
    }
  }
  return rm;
}

MatrixView newView(unsigned rows, unsigned cols) {
    MatrixView mv;
    mv.rows = rows;
    mv.cols = cols;
    mv.data = new float[rows * cols];
    return mv;
}

void releaseView(MatrixView &view) {
    delete[] view.data;
}
